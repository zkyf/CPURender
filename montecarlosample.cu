#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <>
#include <helper_functions.h>
#include <hip/hip_runtime_api.h>
#include <stdio.h>
#include "cugeometry.h"

static CudaGeometry* dev_geos = nullptr;
static int n = 0;

extern "C"
__global__ void IntersectGeo(CudaGeometry* geolist, int n, CudaRay ray)
{
  if(blockIdx.x<n)
  {
    CudaVertex hp = ray.IntersectGeo(geolist[blockIdx.x]);
    if(hp.valid)
    {
      printf("CUDA # %d : hit geo with index # %d\n", blockIdx.x, geolist[blockIdx.x].index);
    }
  }
}

extern "C"
__global__ void testKernel(int val)
{
    printf("[%d, %d]:\t\tValue is:%d\n",\
            blockIdx.y*gridDim.x+blockIdx.x,\
            threadIdx.z*blockDim.x*blockDim.y+threadIdx.y*blockDim.x+threadIdx.x,\
            val);
}

extern "C"
void CudaIntersect(const CudaRay& ray)
{
  IntersectGeo<<n, 1>>(dev_geos, n, ray);
}

extern "C"
void CudaInit(CudaGeometry* geos)
{
  if(!geos)
  {
    return;
  }

  if(!dev_geos)
  {
    hipFree(dev_geos);
  }

  n = input.size();
  for(int i=0; i<input; i++)
  {
    CudaGeometry newg;
    newg.index=i;
    newg.diffuse = CudaVec(input[i].diffuse.r, input[i].diffuse.g, input[i].diffuse.b);
    newg.emission = CudaVec(input[i].emission.r, input[i].emission.g, input[i].emission.b);
    newg.specular = CudaVec(input[i].specular.r, input[i].specular.g, input[i].specular.b);
    newg.reflectr = input[i].reflectr;
    newg.refractr = input[i].refractr;

    for(int j=0; j<input[i].vecs.size(); j++)
    {
      newg.vecs[j].p = input[i].vecs[j].p;
      newg.vecs[j].n = input[i].vecs[j].n;
      newg.vecs[j].geo = i;
    }
  }

  hipMalloc((void**)&dev_geos, sizeof(CudaGeometry)*n);
  hipMemcpy(dev_geos, geos, sizeof(CudaGeometry)*n, hipMemcpyHostToDevice);
}

extern "C"
void CudaEnd()
{
  if(geos)
  {
    delete[] geos;
    geos = nullptr;
    hipFree(dev_geos);
  }
}
