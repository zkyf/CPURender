#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <>
#include <helper_functions.h>
#include <hip/hip_runtime_api.h>
#include <stdio.h>
#include <ctime>
using namespace std;
//#include "cugeometry.h"

#define M_PI (3.1415926)

#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess)
   {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}

__device__ int CudaSudoRandomInt(int& seed)
{
  seed++;
//  seed = (seed * 32983 + 92153) % 19483;
//  int result = seed % 19483;
  return seed;
}

__device__ double CudaSudoRandom(int & seed)
{
  int t = CudaSudoRandomInt(seed);
  return (double)(t % 19483) / 19483;
}

struct CudaVec;
struct CudaVec4;
struct CudaVertex;
struct CudaGeometry;
struct CudaRay;

struct CudaVec
{
  double x, y, z;

  __host__ __device__ CudaVec(double _x=0.0, double _y=0.0, double _z=0.0) : x(_x), y(_y), z(_z) {}

  __host__ __device__ double Dot(const CudaVec& b)    const { return x*b.x+y*b.y+z*b.z; }
  __host__ __device__ CudaVec Cross(const CudaVec& b) const { return CudaVec(y*b.z-z*b.y, z*b.x-x*b.z, x*b.y-y*b.x); }
  __host__ __device__ double Length()                 const { return sqrt(x*x+y*y+z*z); }
  __host__ __device__ CudaVec Normalized()            const { return CudaVec(x/Length(), y/Length(), z/Length()); }
  __host__ __device__ void Print()                    const { printf("CudaVec(%lf, %lf, %lf)", x, y, z); }

  __host__ __device__ void operator=(const CudaVec& b) { x=b.x; y=b.y; z=b.z; }
};

struct CudaVec4
{
  double x, y, z, w;

  __host__ __device__ CudaVec4(double _x=0.0, double _y=0.0, double _z=0.0, double _w=0.0) : x(_x), y(_y), z(_z), w(_w) {}
  __host__ __device__ CudaVec4(CudaVec v3, double _w=0.0) : x(v3.x), y(v3.y), z(v3.z), w(_w) {}

  __host__ __device__ double   Dot(const CudaVec4& b) const { return x*b.x+y*b.y+z*b.z+w*b.w; }
  __host__ __device__ double   Length()               const { return sqrt(x*x+y*y+z*z+w*w); }
  __host__ __device__ CudaVec4 Normalized()           const { return CudaVec4(x/Length(), y/Length(), z/Length(), w/Length()); }
  __host__ __device__ CudaVec  Vec3()                 const { return CudaVec(x, y, z); }
  __host__ __device__ void     Print()                const { printf("CudaVe4c(%lf, %lf, %lf, %lf)", x, y, z, w); }
};

__host__ __device__ CudaVec operator+(const CudaVec& a, const CudaVec& b)    { return CudaVec(a.x+b.x, a.y+b.y, a.z+b.z); }
__host__ __device__ CudaVec operator-(const CudaVec& a, const CudaVec& b)    { return CudaVec(a.x-b.x, a.y-b.y, a.z-b.z); }
__host__ __device__ CudaVec operator*(const CudaVec& a, const CudaVec& b)    { return CudaVec(a.x*b.x, a.y*b.y, a.z*b.z); }
__host__ __device__ CudaVec operator*(const CudaVec& a, const double& b)     { return CudaVec(a.x*b, a.y*b, a.z*b); }
__host__ __device__ CudaVec operator*(const double& b, const CudaVec& a)     { return CudaVec(a.x*b, a.y*b, a.z*b); }
__host__ __device__ CudaVec operator/(const CudaVec& a, const double& b)     { return CudaVec(a.x/b, a.y/b, a.z/b); }

__host__ __device__ CudaVec4 operator+(const CudaVec4& a, const CudaVec4& b) { return CudaVec4(a.x+b.x, a.y+b.y, a.z+b.z, a.w+b.w); }
__host__ __device__ CudaVec4 operator-(const CudaVec4& a, const CudaVec4& b) { return CudaVec4(a.x-b.x, a.y-b.y, a.z-b.z, a.w-b.w); }
__host__ __device__ CudaVec4 operator*(const CudaVec4& a, const double& b)   { return CudaVec4(a.x*b, a.y*b, a.z*b, a.w*b); }
__host__ __device__ CudaVec4 operator*(const double& b, const CudaVec4& a)   { return CudaVec4(a.x*b, a.y*b, a.z*b, a.w*b); }
__host__ __device__ CudaVec4 operator/(const CudaVec4& a, const double& b)   { return CudaVec4(a.x/b, a.y/b, a.z/b, a.w/b); }

struct CudaVertex
{
  CudaVec4 p;
  CudaVec4 n;
  bool valid;
  int geo;

  __host__ __device__ CudaVertex(bool v=true) : valid(v), p(), n(), geo(-1) {}
  __host__ __device__ static CudaVertex Intersect(const CudaVertex& a, const CudaVertex& b, double r)
  {
    CudaVertex result;
    result.n = a.n*r+b.n*(1-r);
    result.p = a.p*r+b.p*(1-r);

    result.valid = a.valid&&b.valid;
    result.geo = a.geo;

    return result;
  }
  __host__ __device__ void operator=(const CudaVertex& b)
  {
    p=b.p;
    n=b.n;
    valid=b.valid;
    geo=b.geo;
  }
};

struct CudaGeometry
{
  int index;
  bool selected = false;
  CudaVertex vecs[3];
  CudaVec diffuse = CudaVec(1.0, 1.0, 1.0);
  CudaVec specular = CudaVec();
  CudaVec emission = CudaVec();
  double reflectr = 0.0;
  double refractr = 0.0;
  __device__ CudaVertex Sample(int& seed, double* randNum, int randN) const
  {
    // to do
    CudaVertex result = vecs[0];
    for(int i=1; i<3; i++)
    {
      result = CudaVertex::Intersect(result, vecs[i], randNum[CudaSudoRandomInt(seed)%randN]);
    }
    return result;
  }
  __host__ __device__ CudaVec Normal() const
  {
    return (vecs[2].p-vecs[1].p).Vec3().Cross(vecs[0].p.Vec3()-vecs[1].p.Vec3()).Normalized();
  }

  __host__ __device__ CudaVec4 Plane() const
  {
    CudaVec4 n(Normal(), 0.0);
    n.w = (-n.Dot(vecs[0].p));
    return n;
  }
  __host__ __device__ void Print() const
  {
    printf("CudaGeometry # %d\n", index);
    for(int i=0; i<3; i++)
    {
      printf("v#%d p=", i); vecs[i].p.Print();
      printf(", n="); vecs[i].n.Print();
      printf("\n");
    }
  }
};

struct CudaRay
{
  CudaVec4 o;
  CudaVec4 n;

  __host__ __device__ void IntersectGeo(const CudaGeometry& geo, CudaVertex& result) const
  {
//    printf("CudaRay IntersectGeo # %d: o=(%lf, %lf, %lf), n=(%lf, %lf, %lf)\n", geo.index, o.x, o.y, o.z, n.x, n.y, n.z);
//    geo.Print();
    CudaVec4 pi = geo.Plane();
    const CudaVertex c = geo.vecs[0];
    const CudaVertex a = geo.vecs[1];
    const CudaVertex b = geo.vecs[2];

//    printf("CudaRay IntersectGeo: o=(%lf, %lf, %lf), n=(%lf, %lf, %lf), geo # %d  pi=(%lf, %lf, %lf, %lf)\n", o.x, o.y, o.z, n.x, n.y, n.z, geo.index, pi.x, pi.y, pi.z, pi.w);
    double pn = pi.Dot(n);

    if(fabs(pn)<1e-3)
    {
//      printf("CudaRay IntersectGeo: o=(%lf, %lf, %lf), n=(%lf, %lf, %lf), parallel geo # %d  pi=(%lf, %lf, %lf, %lf)\n", o.x, o.y, o.z, n.x, n.y, n.z, geo.index, pi.x, pi.y, pi.z, pi.w);
      result.valid=false;
      return;
    }

    double r = -pi.Dot(o)/pn;

//    printf("r=%lf, pn=%lf\n", r, pn);
    if(r<0)
    {
      result.valid=false;
      return;
    }
    CudaVec4 p=r*n+o; // intersection point with the plane

//    printf("CudaRay IntersectGeo: o=(%lf, %lf, %lf), n=(%lf, %lf, %lf), hit geo # %d @ p=(%lf, %lf, %lf)\n", o.x, o.y, o.z, n.x, n.y, n.z, geo.index, p.x, p.y, p.z);

    if((c.p-p).Vec3().Length()<1e-3)
    {
      result.valid=false;
      return;
    }

    CudaVec4 cp = p-c.p;
    cp.w =0;
    CudaVec4 cpn = cp.Normalized();
    CudaVec4 ca = a.p - c.p;
    CudaVec4 cb = b.p - c.p;
    CudaVec4 cd = ca.Dot(cpn)*cpn;
    CudaVec4 ce = cb.Dot(cpn)*cpn;

//    printf("cp="); cp.Print(); printf("\n");
//    printf("ca="); ca.Print(); printf("\n");
//    printf("cb="); cb.Print(); printf("\n");
//    printf("cd="); cd.Print(); printf("\n");
//    printf("ce="); ce.Print(); printf("\n");

    if((ca-cd).Dot(cb-ce)>0)
    {
      result.valid=false;
      return;
    }
    double rb, ra;
    if((cb-ce).Length()>1e-3)
    {
      rb = (ca-cd).Length()/(cb-ce).Length();
      if(rb<0)
      {
        result.valid=false;
        return;
      }
      rb = rb/(1+rb);
      ra = 1-rb;
    }
    else
    {
      ra = 0;
      rb = 1.0;
    }
    CudaVec4 f = rb*b.p+ra*a.p;
//    if((f-c.p).Length()<1e-3)
//    {
//      result.valid=false;
//      return;
//    }
    double rc = 1-cp.Length()/(f-c.p).Length();
    if(cp.Dot(f-c.p)<0)
    {
      result.valid=false;
      return;
    }

//    printf("ra=%lf rb=%lf rc=%lf\n", ra, rb, rc);

    if(ra<0 || rb<0 || rc<0 || ra>1 || rb>1 || rc>1)
    {
      result.valid=false;
      return;
    }
    else
    {
      CudaVertex vf = CudaVertex::Intersect(a, b, ra);
      CudaVertex vp = CudaVertex::Intersect(c, vf, rc);
      vp.geo = geo.index;
      vp.valid=true;
      result = vp;
    }
  }

  __host__ __device__ void Print() const { printf("CudaRay: o=(%lf, %lf, %lf) , n=(%lf, %lf, %lf)\n", o.x, o.y, o.z, n.x, n.y, n.z); }
};

struct CudaKdTree
{
  struct Node
  {
    char axis;

    float xmin = 1e20;
    float ymin = 1e20;
    float zmin = 1e20;

    float xmax = -1e20;
    float ymax = -1e20;
    float zmax = -1e20;

    float value;
    int depth;
    Node* left;
    Node* right;
    Node* parent;
    int n;
    int* gl;
    CudaGeometry* geolist;

    __host__ __device__ Node(Node* p = nullptr) : parent(p) {}
    __host__ __device__ ~Node()
    {
      if(left)
      {
        left->~Node();
      }
      if(right)
      {
        right->~Node();
      }
    }

    __host__ __device__ bool Inside(CudaVec p)
    {
      if(p.x>=xmin && p.x<=xmax && p.y>=ymin && p.y<=ymax && p.z>=zmin && p.z<=zmax)
      {
        return true;
      }
      else
      {
        return false;
      }
    }

    __host__ __device__ void Split()
    {
      if(depth==0)
      {
        for(int gid=0; gid<n; gid++)
        {
          int geo = gl[gid];
          for(int vid=0; vid<3; vid++)
          {
            if(xmin>geolist[geo].vecs[vid].p.x) xmin = geolist[geo].vecs[vid].p.x;
            if(ymin>geolist[geo].vecs[vid].p.y) ymin = geolist[geo].vecs[vid].p.y;
            if(zmin>geolist[geo].vecs[vid].p.z) zmin = geolist[geo].vecs[vid].p.z;

            if(xmax<geolist[geo].vecs[vid].p.x) xmax = geolist[geo].vecs[vid].p.x;
            if(ymax<geolist[geo].vecs[vid].p.y) ymax = geolist[geo].vecs[vid].p.y;
            if(zmax<geolist[geo].vecs[vid].p.z) zmax = geolist[geo].vecs[vid].p.z;
          }
        }
      }

      if(n<=1) { return; }

      float xd = 0.0, xmean = 0.0;
      float yd = 0.0, ymean = 0.0;
      float zd = 0.0, zmean = 0.0;
      int vcount = 0;

      for(int gid=0; gid<n; gid++)
      {
        int geo = gl[gid];
        for(int vid=0; vid<3; vid++)
        {
          if(!Inside(geolist[geo].vecs[vid].p.Vec3())) continue;
          vcount++;
          xmean += geolist[geo].vecs[vid].p.x;
          ymean += geolist[geo].vecs[vid].p.y;
          zmean += geolist[geo].vecs[vid].p.z;
        }
      }

      for(int gid=0; gid<n; gid++)
      {
        int geo = gl[gid];
        for(int vid=0; vid<3; vid++)
        {
          if(!Inside(geolist[geo].vecs[vid].p.Vec3())) continue;
          xd += pow(geolist[geo].vecs[vid].p.x-xmean, 2);
          yd += pow(geolist[geo].vecs[vid].p.y-ymean, 2);
          zd += pow(geolist[geo].vecs[vid].p.z-zmean, 2);
        }
      }

      xd/=vcount; yd/=vcount; zd/=vcount;

    }

  }; // end of CudaKdTree::Node
}; // End of CudaKdTree

#define SampleNum (32)
#define RayTraceLength (5)

static CudaGeometry* dev_geos = nullptr;
static int n = 0;
static FILE* cudaOutput;
static double* hits = nullptr;
static double* dev_hits = nullptr;
static double* dev_randNum = nullptr;
static int randN = 0;
static int* lightGeoList = nullptr;
static int* dev_lightGeoList = nullptr;
static int ln = 0;

__global__ void IntersectGeo(CudaGeometry* geolist, int n, CudaRay ray, double* dev_hits)
{
//  printf("\n");
//  printf("CUDA block (%d, %d, %d) thread (%d %d %d) : ray\n", blockIdx.x, blockIdx.y, blockIdx.z, threadIdx.x, threadIdx.y, threadIdx.z);
  if(blockIdx.x<n)
  {
    CudaVertex hp;
    ray.IntersectGeo(geolist[blockIdx.x], hp);
//    ray.Print();
//    printf("\n");
    if(hp.valid)
    {
//      printf("CUDA # %d : hit geo with index # %d\n", blockIdx.x, geolist[blockIdx.x].index);
      geolist[hp.geo].selected = !geolist[hp.geo].selected;
      dev_hits[blockIdx.x]=(hp.p-ray.o).Vec3().Length();
    }
    else
    {
//      printf("CUDA # %d : NOT hit geo with index # %d\n", blockIdx.x, geolist[blockIdx.x].index);
      dev_hits[blockIdx.x]=-1;
    }
  }
//  printf("\n");
}

__device__ void CudaMonteCarloSample(CudaGeometry* geolist, int n, int* lightGeoList, int ln, CudaVertex o, CudaRay i, double* randNum, int randN, CudaVec& result, int index, bool debuginfo = false)
{
  if(!o.valid) return;
  o.n.w=0;
  o.n = o.n.Normalized();


  CudaVec currentColor=geolist[o.geo].diffuse;

  bool haslight=geolist[o.geo].emission.Length()>0.1;
  CudaRay currentIn = i;

  if(haslight)
  {
    result = geolist[o.geo].emission;
    return;
  }

  for(int level=0; level<RayTraceLength; level++)
  {
    if(o.n.Dot(currentIn.n)>0) o.n = -1.0*o.n;
    if(debuginfo)
    {
      printf("======================================\n");
      printf("Current Geo # %d: ", o.geo);
      printf("reflect %.6lf refract %.6lf\n", geolist[o.geo].reflectr, geolist[o.geo].refractr);
      printf("Current Level: %d, index = %d\n", level, index);
      printf("CurrentO = "); o.p.Print(); printf("\n");
      printf("CurrentColor = "); currentColor.Print();
      printf("\n");
    }

    CudaRay ray;
    ray.o=o.p;

    if(geolist[o.geo].reflectr>1e-3)
    {
      ray.n = (currentIn.n-2*o.n*(currentIn.n.Dot(o.n))).Normalized();
    }
    else
    {
      float sin2theta=randNum[CudaSudoRandomInt(index)%randN]; // sin^2(theta)
      float sintheta=sqrt(sin2theta);
      float phi = randNum[CudaSudoRandomInt(index)%randN]*2*M_PI;
      if(debuginfo)
      {
        printf("sin2theta = %.6lf, phi = %.6lf, index = %d\n", sin2theta, phi, index);
      }
      CudaVec w = o.n.Vec3();
      CudaVec u = (fabs(w.x)>0.1?CudaVec(0, 1):CudaVec(1)).Cross(w);
      CudaVec v = w.Cross(u);
      ray.n = CudaVec4(sintheta*cos(phi)*u+v*sintheta*sin(phi)+w*sqrt(1-sin2theta), 0);
      if(debuginfo)
      {
        printf("w = "); w.Print(); printf(", u = "); u.Print(); printf(", v = "); v.Print(); printf("\n");
        printf("generated ray: "); ray.Print(); printf("\n");
      }
    }

    double mind=1e20;
    CudaVertex minp(false);

    for(int i=0; i<n; i++)
    {
      CudaVertex hp;
      ray.IntersectGeo(geolist[i], hp);
      if(hp.valid)
      {
        if((hp.p-o.p).Length()>1e-3 && (hp.p-o.p).Length()<mind)
        {
          mind=(hp.p-o.p).Length();
          minp=hp;
        }
      }
    }

    if(minp.valid)
    {
      if(debuginfo)
      {
        printf("Ray hit geo # %d @", minp.geo);
        minp.p.Print();
        printf("\n");
      }

      if(geolist[minp.geo].emission.Length()>0.1)
      {
        float r = 1.0/(minp.p-o.p).Length();
        r = r*r;
        if(r>1/fabs(ray.n.Dot(o.n))) r=1/fabs(ray.n.Dot(o.n));
        currentColor = currentColor * geolist[minp.geo].emission * fabs(ray.n.Dot(o.n)) * r * 3;
        if(debuginfo)
        {
          printf("got light color: "); geolist[minp.geo].emission.Print(); printf("\n");
          printf("current color = "); currentColor.Print(); printf("\n");
        }
        haslight=true;
      }
      else
      {
        float r = 1.0/(minp.p-o.p).Length();
        r = r*r;
        if(r>1) r=1;
        currentColor = currentColor * geolist[minp.geo].diffuse * fabs(ray.n.Dot(o.n)) * r * 0.9;
        if(debuginfo)
        {
          printf("got diffuse color: "); geolist[minp.geo].diffuse.Print(); printf("\n");
          printf("current color = "); currentColor.Print(); printf("\n");
        }
      }
    }
    else
    {
      if(debuginfo)
      {
        printf("ray hit nothing\n");
        printf("======================================\n\n");
      }
      break;
    }

    if(debuginfo)
    {
      printf("======================================\n\n");
    }

    o = minp;
    currentIn = ray;
  }

  if(haslight)
  {
    if(debuginfo)
    {
      printf("FINAL: haslight "); currentColor.Print(); printf("\n");
    }
    result = currentColor;
    return;
  }
  else
  {
//    result = currentColor;
    if(debuginfo)
    {
      printf("FINAL: NOT haslight "); currentColor.Print(); printf("\n");
      printf("# %d lightGeo in total\n", ln);
    }
    result = CudaVec(0, 0, 0);
    if(o.valid)
    {
      if(geolist[o.geo].reflectr>1e-3)
      {
        return;
      }

      int hitcount=0;
      CudaVec lightColor;
      for(int ii=0; ii<ln; ii++)
      {
        if(debuginfo)
        {
          printf("\n\ntest lightGeo # %d\n", lightGeoList[ii]);
        }
//        for(int j=0; j<; j++)
        {
          CudaVertex v = geolist[lightGeoList[ii]].Sample(index, randNum, randN);
          if(debuginfo)
          {
//            printf("  sample vertex # %d\n", j);
            printf("  d = %.6lf, ", (v.p-o.p).Length()); v.p.Print(); printf("\n");
          }
          CudaRay ray; ray.o = o.p; ray.n = CudaVec4((v.p.Vec3()-o.p.Vec3()).Normalized(), 0.0);
          bool visible = true;
          int lighthit = lightGeoList[ii];
          double d = (v.p-o.p).Length();
          for(int k=0; k<n; k++)
          {
            if(k==lightGeoList[ii]) continue;
            CudaVertex ir;
            ray.IntersectGeo(geolist[k], ir);
            if(ir.valid && (ir.p-o.p).Length()<d && (ir.p-o.p).Length()>1e-3)
            {
              v = ir;
              d = (ir.p-o.p).Length();
              if(debuginfo)
              {
                printf("  "); ray.Print();
                printf("  geo # %d got hit @ ", k); ir.p.Print(); printf(" d = %lf emission = %.6lf\n\n", d, geolist[k].emission.Length());
              }
              if(geolist[k].emission.Length()>0.1)
              {
                lighthit = k;
                visible = true;
              }
              else
              {
                visible = false;
              }
            }
          }
          if(visible)
          {
            hitcount++;
            float r = 1.0/d*d;
            if(r>1/fabs(ray.n.Dot(o.n))) r=1/fabs(ray.n.Dot(o.n));
            lightColor = lightColor + geolist[lighthit].emission * fabs(ray.n.Dot(o.n))*r * 3;
            if(debuginfo)
            {
              printf("lightgeo # %d visible", lighthit); (geolist[lighthit].emission * fabs(ray.n.Dot(o.n))).Print(); printf("\n");
              printf("now light color = "); lightColor.Print(); printf("\n");
            }
          }
        }
      }
      if(hitcount>0)
      {
        result = currentColor * lightColor / hitcount;
        if(debuginfo)
        {
          printf("\n\nFinal result color = "); result.Print(); printf("\n");
        }
      }
    }
    return;
  }
}

__device__ CudaRay GetRay(int xx, int yy, int width, int height, CudaVec camera, CudaVec up, CudaVec forward, CudaVec right)
{
  float vh = M_PI/3;
  float vw = vh/height * width;
  float vhp = vh/height;
  float vwp = vw/width;

  CudaRay ray;
  ray.o = CudaVec4(camera, 1.0);
  ray.n = CudaVec4((forward+right*tan(vwp*xx-vw/2)+up*tan(vhp*(height - yy - 1)-vh/2)).Normalized(), 0.0);

  return ray;
}

__global__ void debug_GetRay(int xx, int yy, int width, int height, CudaVec camera, CudaVec up, CudaVec forward, CudaVec right, CudaRay* result)
{
  printf("debug_GetRay\n");
  float vh = M_PI/3;
  float vw = vh/height * width;
  float vhp = vh/height;
  float vwp = vw/width;

  CudaRay ray;
  ray.o = CudaVec4(camera, 1.0);
  ray.n = CudaVec4((forward+right*tan(vwp*xx-vw/2)+up*tan(vhp*(height - yy - 1)-vh/2)).Normalized(), 0.0);

  printf("ray.n=(%lf, %lf, %lf)\n", ray.n.x, ray.n.y, ray.n.z);
  *result = ray;
}

__global__ void CudaMonteCarloRender(CudaGeometry* geolist, int n, int* lightGeoList, int ln, int w, int h, CudaVec camera, CudaVec up, CudaVec forward, CudaVec right, CudaVec* buffer, double* randNum, int randN, bool debuginfo = false)
{
  __shared__ CudaVec tResults[SampleNum];
  if(n==0) return;
  int xx=blockIdx.x;
  int yy=blockIdx.y;
  int tid=threadIdx.x;
  tResults[tid] = CudaVec(0, 0, 0);

  if(xx<0 || xx>=w || yy<0 || yy>=h) return;
  int index = xx+yy*w;
  int seed = index*blockDim.x+threadIdx.x;
  CudaRay ray = GetRay(xx, yy, w, h, camera, up, forward, right);
//  buffer[index] = CudaVec(0, 0, 0);

//  for(int sp=0; sp<SampleNum; sp++)
  {
    double mind=1e20;
    CudaVertex minp(false);

//    buffer[index] = buffer[index]+(ray.n.Vec3()+CudaVec(1.0, 1.0, 1.0))/2;

    for(int gi=0; gi<n; gi++)
    {
      CudaVertex hp(false);
      CudaGeometry geo = geolist[gi];
      ray.IntersectGeo(geo, hp);
      if(hp.valid==true)
      {
        double d=(hp.p.Vec3()-camera).Length();
        if(d>1e-3 && d<mind)
        {
          mind=d;
          minp=hp;
        }
      }
    }

    if(minp.valid)
    {
//      minp.valid=false;
      CudaVec result;
      CudaMonteCarloSample(geolist, n, lightGeoList, ln, minp, ray, randNum, randN, result, seed * 17, debuginfo);
      tResults[tid] = result;
    }
  }

  if(tid==0)
  {
    int count=1;
    for(int i=1; i<blockDim.x; i++)
    {
      if(tResults[tid].Length()>1e-3)
      tResults[tid] = tResults[tid]+tResults[i];
      count++;
    }
    buffer[index] = tResults[tid]/count;
  }
  __syncthreads();

  return;
}
__global__ void CudaDivide(int w, int h, CudaVec* buffer)
{
  int xx=blockIdx.x;
  int yy=blockIdx.y;
  if(xx<0 || xx>=w || yy<0 || yy>=h) return;
  int index = xx+yy*gridDim.x;
  buffer[index] = buffer[index]/4;
}

__global__ void debug_MonteCarloRender(CudaGeometry* geolist, int n, int* lightGeoList, int ln, int w, int h, CudaVec camera, CudaVec up, CudaVec forward, CudaVec right, CudaVec* buffer, double* randNum, int randN, bool debuginfo = false, int _xx=-1, int _yy=1)
{
  int xx = _xx;
  int yy = _yy;
  int index = xx+yy*gridDim.x;
  if(xx<0 || xx>=w || yy<0 || yy>=h) return;
  CudaRay ray = GetRay(xx, yy, w, h, camera, up, forward, right);
  buffer[index] = CudaVec(0, 0, 0);

  printf("debug_MonteCarloRender @ %d %d\n", xx, yy);

  for(int sp=0; sp<1; sp++)
  {
    double mind=1e20;
    CudaVertex minp(false);

//    buffer[index] = buffer[index]+(ray.n.Vec3()+CudaVec(1.0, 1.0, 1.0))/2;

    for(int gi=0; gi<n; gi++)
    {
      CudaVertex hp(false);
      CudaGeometry geo = geolist[gi];
      ray.IntersectGeo(geo, hp);
      if(hp.valid==true)
      {
        double d=(hp.p.Vec3()-camera).Length();
        if(d>1e-3 && d<mind)
        {
          mind=d;
          minp=hp;
        }
      }
    }

    if(minp.valid)
    {
//      minp.valid=false;
      CudaVec result;
      CudaMonteCarloSample(geolist, n, lightGeoList, ln, minp, ray, randNum, randN, result, index, debuginfo);
      buffer[index] = buffer[index] + result;
    }
  }

//  buffer[index] = buffer[index]/SampleNum;

  return;
}

extern "C" void CudaMonteCarloSampleTest(int xx, int yy, int w, int h, CudaVec camera, CudaVec up, CudaVec forward, CudaVec right)
{
  float elapsed=0;
  hipEvent_t start, stop;
  CudaVec* dev_buffer;

  gpuErrchk( hipMalloc((void**)&dev_buffer, sizeof(CudaVec)*(w*h+10)));

  printf("start cuda render\n");
  fflush(stdout);
  hipEventCreate(&start);
  hipEventCreate(&stop);
  hipEventRecord(start, 0);
//  for(int i=0; i<4; i++)
//  {
    debug_MonteCarloRender<<<1, 1>>>(dev_geos, n, dev_lightGeoList, ln, w, h, camera, up, forward, right, dev_buffer, dev_randNum, randN, true, xx, yy);
//  }
//  CudaDivide<<<dim3(w, h), 1>>>(w, h, dev_buffer);
  hipEventRecord(stop, 0);
  hipEventSynchronize (stop);
  hipEventElapsedTime(&elapsed, start, stop);
  hipEventDestroy(start);
  hipEventDestroy(stop);
  printf("The elapsed time in gpu was %.2f ms", elapsed);
  fflush(stdout);
  printf("end cuda render\n");
  fflush(stdout);

//  gpuErrchk( hipPeekAtLastError() );
//  gpuErrchk( hipDeviceSynchronize() );

  gpuErrchk( hipFree(dev_buffer));
}

extern "C"
void CudaInit(CudaGeometry* geos, int _n, int* lightList, int _ln, double* h, double* randNum, int _randN)
{
//  cudaOutput = fopen("cudaoutput.txt", "w");
  hits=h;
  printf("cuda part init %d\n", _n);
  fflush(stdout);
  if(!geos)
  {
    return;
  }

  if(!dev_geos)
  {
    gpuErrchk(hipFree(dev_geos));
    gpuErrchk(hipFree(dev_hits));
    gpuErrchk(hipFree(dev_randNum));
    gpuErrchk(hipFree(dev_lightGeoList));
  }

  n = _n;
  randN = _randN;
  ln = _ln;

  printf("CUDA: sizeof(CudaGeometry)=%d, sizeof(CudaVec)=%d, n=%d\n", sizeof(CudaGeometry), sizeof(CudaVec), n);

  gpuErrchk( hipMalloc((void**)&dev_geos, sizeof(CudaGeometry)*n));
  gpuErrchk( hipMalloc((void**)&dev_hits, sizeof(double)*n));
  gpuErrchk( hipMalloc((void**)&dev_lightGeoList, sizeof(int)*ln));
  gpuErrchk( hipMalloc((void**)&dev_randNum, sizeof(double)*randN));
  gpuErrchk( hipMemcpy(dev_geos, geos, sizeof(CudaGeometry)*n, hipMemcpyHostToDevice));
  gpuErrchk( hipMemcpy(dev_randNum, randNum, sizeof(double)*randN, hipMemcpyHostToDevice));
  gpuErrchk( hipMemcpy(dev_lightGeoList, lightList, sizeof(int)*ln, hipMemcpyHostToDevice));
  gpuErrchk( hipPeekAtLastError() );
  fflush(stdout);
}

extern "C"
void CudaEnd()
{
  if(dev_geos)
  {
    gpuErrchk(hipFree(dev_geos));
    gpuErrchk(hipFree(dev_hits));
    gpuErrchk(hipFree(dev_randNum));
    gpuErrchk(hipFree(dev_lightGeoList));
  }
}

extern "C" void CudaRender(int w, int h, CudaVec camera, CudaVec up, CudaVec forward, CudaVec right, CudaVec* buffer)
{
  float elapsed=0;
  hipEvent_t start, stop;
  CudaVec* dev_buffer;
  memset(buffer, 0, sizeof(CudaVec)*w*h);
  gpuErrchk( hipMalloc((void**)&dev_buffer, sizeof(CudaVec)*(w*h+10)));

  printf("start cuda render\n");
  printf("buffer.size=%d %d\n", sizeof(buffer), sizeof(buffer)/sizeof(CudaVec));
  fflush(stdout);
  hipEventCreate(&start);
  hipEventCreate(&stop);
  hipEventRecord(start, 0);
//  for(int i=0; i<4; i++)
//  {
    CudaMonteCarloRender<<<dim3(w, h), SampleNum>>>(dev_geos, n, dev_lightGeoList, ln, w, h, camera, up, forward, right, dev_buffer, dev_randNum, randN);
//  }
//  CudaDivide<<<dim3(w, h), 1>>>(w, h, dev_buffer);
  hipEventRecord(stop, 0);
  hipEventSynchronize (stop);
  hipEventElapsedTime(&elapsed, start, stop);
  hipEventDestroy(start);
  hipEventDestroy(stop);
  printf("The elapsed time in gpu was %.2f ms", elapsed);
  fflush(stdout);
  printf("end cuda render\n");
  fflush(stdout);

//  gpuErrchk( hipPeekAtLastError() );
//  gpuErrchk( hipDeviceSynchronize() );

  clock_t begin = clock();
  gpuErrchk( hipMemcpy(buffer, dev_buffer, sizeof(CudaVec)*(w*h+10), hipMemcpyDeviceToHost));
  clock_t end = clock();
  double elapsed_secs = double(end - begin) / CLOCKS_PER_SEC;
  printf("hipMemcpy costs %.6lf secs.\n", elapsed_secs);
  fflush(stdout);
  gpuErrchk( hipFree(dev_buffer));
}

extern "C" void CudaGetRayTest(int xx, int yy, int w, int h, CudaVec camera, CudaVec up, CudaVec forward, CudaVec right)
{
  printf("CudaGetRayTest: %d %d %d %d\n", xx, yy, w, h);
  camera.Print(); up.Print(); forward.Print(); right.Print();
  printf("\n");
  CudaRay result;
  CudaRay* dev_result;

  hipMalloc((void**)&dev_result, sizeof(CudaRay));

  debug_GetRay<<<1, 1>>>(xx, yy, w, h, camera, up, forward, right, dev_result);
//  hipMemcpy(&result, dev_result, sizeof(CudaRay), hipMemcpyDeviceToHost);
  result.Print();
  ((result.n.Vec3()+CudaVec(1.0, 1.0, 1.0))/2).Print();
  printf("\n");

  fflush(stdout);
}

extern "C" void CudaIntersect(CudaRay ray)
{
  printf("CudaIntersect n=%d\n", n);
  ray.Print();
  fflush(stdout);
  IntersectGeo<<<n, 1>>>(dev_geos, n, ray, dev_hits);
  hipMemcpy(hits, dev_hits, sizeof(double) * n, hipMemcpyDeviceToHost);
  for(int i=0; i<n; i++)
  {
    if(hits[i] >=0 )
    {
//      printf("geo #%d hit\n", i);
    }
    else
    {
//      printf("geo #%d NOT hit\n", i);
    }
  }
//  printf("cuda part ended");
  fflush(stdout);
}
